///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
	return (a + (b - 1)) / b;
}


// complex math functions
__device__
float2 conjugate(float2 arg)
{
	return make_float2(arg.x, -arg.y);
}

__device__
float2 complex_exp(float arg)
{
	return make_float2(cosf(arg), sinf(arg));
}

__device__
float2 complex_add(float2 a, float2 b)
{
	return make_float2(a.x + b.x, a.y + b.y);
}

__device__
float2 complex_mult(float2 ab, float2 cd)
{
	return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

// generate wave heightfield at time t based on initial heightfield and dispersion relationship
__global__ void generateSpectrumKernel(float2* h0, float2 *ht, unsigned int width, unsigned int height, float t, float patchSize)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = y*width+x;

	// calculate coordinates
	float2 k;
	k.x = HIP_PI_F * x / (float) patchSize;
	k.y = 2.0f * HIP_PI_F * y / (float) patchSize;

	// calculate dispersion w(k)
	float k_len = sqrtf(k.x*k.x + k.y*k.y);
	float w = sqrtf(9.81f * k_len);

	float2 h0_k = h0[i];
	float2 h0_mk = h0[(((height-1)-y)*width)+x];

	float2 h_tilda = complex_add( complex_mult(h0_k, complex_exp(w * t)),
		complex_mult(conjugate(h0_mk), complex_exp(-w * t)) );

	// output frequency-space complex values
	if ((x < width) && (y < height)) {
		ht[i] = h_tilda;
	}
}


// generate slope by partial differences in spatial domain
__global__ void calculateSlopeKernel(float* h, float2 *slopeOut, unsigned int width, unsigned int height)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = y*width+x;

	float2 slope;
	if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1)) {
		slope.x = h[i+1] - h[i-1];
		slope.y = h[i+width] - h[i-width];
	} else {
		slope = make_float2(0.0f, 0.0f);
	}
	slopeOut[i] = slope;
}

extern "C" 
void cudaGenerateSpectrumKernel(float2* d_h0, float2 *d_ht, 
								unsigned int width, unsigned int height, 
								float animTime, float patchSize)
{
	dim3 block(8, 8, 1);
	dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
	generateSpectrumKernel<<<grid, block>>>(d_h0, d_ht, width, height, animTime, patchSize);
}

extern "C"
void cudaCalculateSlopeKernel(  float* hptr, float2 *slopeOut, 
							  unsigned int width, unsigned int height)
{
	dim3 block(8, 8, 1);
	dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
	calculateSlopeKernel<<<grid2, block>>>(hptr, slopeOut, width, height);
}

__global__ void KernelFFTCUDA_H(unsigned char* surface, float *dout, int width, int height, size_t pitch, float t)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	float* pixel;
	float value;

	if (x >= width || y >= height) return;

	pixel = (float*)(surface + y*pitch) + 4*x;
	value = dout[y*width + x];

	pixel[0] = value; // red
	pixel[1] = value; // green
	pixel[2] = value; // blue
	pixel[3] = 1; // alpha
}

extern "C" 
void FFTCUDA_H(void* surface, float* dout, int width, int height, size_t pitch, float t)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3( 16, 16 ); // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3( (width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y );

	KernelFFTCUDA_H<<<Dg,Db>>>( (unsigned char*)surface, dout, width, height, pitch, t );

	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("PerlinCUDA_H() failed to launch error = %d\n", error);
	}
}